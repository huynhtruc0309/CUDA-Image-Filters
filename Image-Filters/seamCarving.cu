#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "seamCarving.h"
#include "hip/hip_runtime.h"

#define PROFILE 1
#define STREAMCOMPACTION 1

#define BLOCK_SIZE 256
const dim3 blockSize(16, 16, 1);

namespace seamCarving {

	void computeFullEnergy(Mat im, unsigned int *ene) {
		Mat energy(im.rows, im.cols, CV_32S, Scalar(195075));

		for (int i = 1; i < im.rows - 1; ++i) {
			uchar* prev = im.ptr<uchar>(i - 1);	
			uchar* curr = im.ptr<uchar>(i);		
			uchar* next = im.ptr<uchar>(i + 1);	

			for (int j = 1; j < im.cols - 1; ++j) {
				int val = 0;
				val += (prev[3 * j] - next[3 * j]) * (prev[3 * j] - next[3 * j]);
				val += (prev[3 * j + 1] - next[3 * j + 1]) * (prev[3 * j + 1] - next[3 * j + 1]);
				val += (prev[3 * j + 2] - next[3 * j + 2]) * (prev[3 * j + 2] - next[3 * j + 2]);

				val += (curr[3 * j + 3] - curr[3 * j - 3]) * (curr[3 * j + 3] - curr[3 * j - 3]);
				val += (curr[3 * j + 4] - curr[3 * j - 2]) * (curr[3 * j + 4] - curr[3 * j - 2]);
				val += (curr[3 * j + 5] - curr[3 * j - 1]) * (curr[3 * j + 5] - curr[3 * j - 1]);

				energy.at<unsigned int>(i, j) = val;
			}
		}
		for (size_t i = 0; i < im.rows; i++) {
			for (size_t j = 0; j < im.cols; j++) {
				ene[(i*im.cols) + j] = (int)energy.at<unsigned int>(i, j);
			}
		}				
	}

	unsigned int getEnergy(unsigned int *energy, unsigned int row, unsigned int col, size_t cols) {
		cout << row << ", " << col << ", " << row*cols + col << endl;
		return energy[row*cols+col];
	}

	vector<uint> findVerticalSeam(unsigned int *energy, size_t rows, size_t cols) {
		vector<uint> seam(rows);
		unsigned int** distTo = new unsigned int*[rows];
		short** edgeTo = new short*[rows];
		for (int i = 0; i < rows; ++i) {
			distTo[i] = new unsigned int[cols];
			edgeTo[i] = new short[cols];
		}

		for (int i = 0; i < rows; ++i) {
			for (int j = 0; j < cols; ++j) {
				if (i == 0)		distTo[i][j] = 0;
				else			distTo[i][j] = numeric_limits<unsigned int>::max();
				edgeTo[i][j] = 0;
			}
		}

		for (int row = 0; row < rows - 1; ++row) {
			for (int col = 0; col < cols; ++col) {
				if (col != 0)
					if (distTo[row + 1][col - 1] > distTo[row][col] + energy[(row + 1) * cols + (col - 1)]) {
						distTo[row + 1][col - 1] = distTo[row][col] + energy[(row + 1) * cols + (col - 1)];
						edgeTo[row + 1][col - 1] = 1;
					}
				if (distTo[row + 1][col] > distTo[row][col] + energy[(row + 1)*cols + col]) {
					distTo[row + 1][col] = distTo[row][col] + energy[(row + 1)*cols + col];
					edgeTo[row + 1][col] = 0;
				}
				if (col != cols-1)
					if (distTo[row + 1][col + 1] > distTo[row][col] + energy[(row+1)*cols + (col +1 )]) {
						distTo[row + 1][col + 1] = distTo[row][col] + energy[(row + 1)*cols + (col + 1)];
						edgeTo[row + 1][col + 1] = -1;
					}
			}
		}

		unsigned int min_index = 0, min = distTo[rows - 1][0];
		for (int i = 1; i < cols; ++i)
		if (distTo[rows - 1][i] < min) {
			min_index = i;
			min = distTo[rows - 1][i];
		}

		seam[rows - 1] = min_index;
		for (int i = rows - 1; i > 0; --i)
			seam[i - 1] = seam[i] + edgeTo[i][seam[i]];
		return seam;
	}

	void removeVerticalSeam(vector<uint> seam, Mat *im) {
		size_t rows = (*im).rows;
		size_t cols = (*im).cols;
		for (int row = 0; row < rows; ++row) {
			for (int col = seam[row]; col < cols - 1; ++col){
				(*im).at<Vec3b>(row, col) = (*im).at<Vec3b>(row, col + 1);
			}
		}
		*im = (*im)(Rect(0, 0, cols - 1, rows));
	}

	Mat cpuCarve(cv::Mat im, int direction, int seams) {
		unsigned char *imgPtr = new unsigned char[im.rows * im.cols * im.channels()];

		unsigned char *cvPtr = im.ptr<unsigned char>(0);
		for (size_t i = 0; i < im.rows * im.cols * im.channels(); ++i) {
			imgPtr[i] = cvPtr[i];
		}

		size_t rows = im.rows;
		size_t cols = im.cols;
		unsigned int *energy = new unsigned int[rows*cols];
		// Horizontal
		if (direction == 0) {
			for (int i = 0; i < seams; ++i) {
				size_t rows = im.rows;
				size_t cols = im.cols;
				transpose(im, im);
				computeFullEnergy(im, energy);
				vector<uint> seam = findVerticalSeam(energy, cols, rows);
				removeVerticalSeam(seam, &im);
				transpose(im, im);
			}
		}
		// Vertical
		else {
			for (int i = 0; i < seams; ++i) {
				size_t rows = im.rows;
				size_t cols = im.cols;
				#if PROFILE
					CpuTimer timer;
					timer.Start();
				#endif
				computeFullEnergy(im, energy);
				#if PROFILE
					timer.Stop();
					printf("Energy, %f, ", timer.Elapsed());
				#endif
				#if PROFILE
					timer.Start();
				#endif
				vector<uint> seam = findVerticalSeam(energy, rows, cols);
				#if PROFILE
					timer.Stop();
					printf("findSeam, %f,", timer.Elapsed());
				#endif
				#if PROFILE
					timer.Start();
				#endif
				removeVerticalSeam(seam,&im);
				#if PROFILE
					timer.Stop();
					printf("SeamRemoval, %f\n", timer.Elapsed());
				#endif
			}
		}
		return im;
	}

	__global__ void computeEnergyGPU(size_t rows, size_t cols, unsigned char *img, unsigned int *energy) {
		int  ny = rows;
		int  nx = cols;
		int2 image_index_2d = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y * blockDim.y) + threadIdx.y);
		int  image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;

		if (image_index_2d.x < nx && image_index_2d.y < ny)	{
			if (image_index_2d.x == 0 || image_index_2d.x == cols - 1 || image_index_2d.y == 0 || image_index_2d.y == rows - 1) {
				energy[image_index_1d] = 195075;
			}
			else {
				int val = 0;
				int i = image_index_2d.y;
				int j = image_index_2d.x;
				val += powf(img[((i - 1)*cols * 3) + (3 * j)] - img[((i + 1)*cols * 3) + (3 * j)], 2);
				val += powf(img[((i - 1)*cols * 3) + ((3 * j) + 1)] - img[((i + 1)*cols * 3) + ((3 * j) + 1)], 2);
				val += powf(img[((i - 1)*cols * 3) + ((3 * j) + 2)] - img[((i + 1)*cols * 3) + ((3 * j) + 2)], 2);

				val += powf(img[(i * cols * 3) + ((3 * j) + 3)] - img[(i * cols * 3) + ((3 * j) - 3)], 2);
				val += powf(img[(i * cols * 3) + ((3 * j) + 4)] - img[(i * cols * 3) + ((3 * j) - 2)], 2);
				val += powf(img[(i * cols * 3) + ((3 * j) + 5)] - img[(i * cols * 3) + ((3 * j) - 1)], 2);

				energy[image_index_1d] = val;
			}
		}
	}

	__global__ void kernMapToBoolean(size_t rows, size_t cols, int *bools, unsigned int *idata) {
		unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= rows*cols*3)
			return;
		bools[index] = 1;
		__syncthreads();
		for (size_t row = 0; row < rows; row++) {
			if (index == (row*cols*3 + idata[row] * 3)) {
				bools[index] = 0;
				bools[index+1] = 0;
				bools[index+2] = 0;
			}
		}
	}

	__global__ void copyElements(int n, int *src, int *dest) {
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= n)
			return;
		dest[index] = src[index];
	}

	__global__ void upSweep(int n, int *idata, int d) {
		int k = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (k >= n)
			return;
		if (k % (d * 2) == (d * 2) - 1) {
			idata[k] += idata[k - d];
		}

	}

	__global__ void downSweep(int n, int *idata, int d) {
		int k = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (k >= n)
			return;
		int temp;
		if (k % (d * 2) == (d * 2) - 1) {
			temp = idata[k - d];
			idata[k - d] = idata[k];
			idata[k] += temp;
		}

	}

	__global__ void makeElementZero(int *data, int index) {
		int k = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index == k) {
			data[k] = 0;
		}
	}

	__global__ void kernScatter(int n, unsigned char *odata, const unsigned char *idata, const int *bools, const int *indices) {
		int index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= n)
			return;
		if (bools[index] == 1)
			odata[indices[index]] = idata[index];
	}

	size_t removeVerticalSeamGPU(size_t rows, size_t cols, unsigned char *dev_idata, unsigned char *dev_odata, unsigned int *dev_seams, int *dev_boolean, int *dev_indices) {

		int count=1;
		int pixelSize = rows*cols * 3;
		int paddedArraySize = 1 << ilog2ceil(pixelSize);

		dim3 fullBlocksPerGrid((pixelSize + BLOCK_SIZE - 1) / BLOCK_SIZE);
		dim3 fullBlocksPerGridPadded((paddedArraySize + BLOCK_SIZE - 1) / BLOCK_SIZE);

		#if PROFILE
			GpuTimer timer;
			timer.Start();
		#endif
		kernMapToBoolean << <fullBlocksPerGrid, BLOCK_SIZE >> >(rows, cols, dev_boolean, dev_seams);

		copyElements << <fullBlocksPerGrid, BLOCK_SIZE >> >(pixelSize, dev_boolean, dev_indices);

		for (int d = 0; d < ilog2ceil(paddedArraySize); d++) {
			upSweep << <fullBlocksPerGridPadded, BLOCK_SIZE >> >(paddedArraySize, dev_indices, 1 << d);
		}

		makeElementZero << <fullBlocksPerGridPadded, BLOCK_SIZE >> >(dev_indices, paddedArraySize - 1);

		for (int d = ilog2ceil(paddedArraySize) - 1; d >= 0; d--) {
			downSweep << <fullBlocksPerGridPadded, BLOCK_SIZE >> >(paddedArraySize, dev_indices, 1 << d);
		}

		kernScatter << <fullBlocksPerGrid, BLOCK_SIZE >> >(pixelSize, dev_odata, dev_idata, dev_boolean, dev_indices);

		#if PROFILE
			timer.Stop();
			printf("Stream Compaction, %f \n", timer.Elapsed());
		#endif

		hipMemcpy(dev_idata, dev_odata, pixelSize*sizeof(unsigned char), hipMemcpyDeviceToDevice);
		hipMemcpy(&count, dev_indices + paddedArraySize - 1, sizeof(int), hipMemcpyDeviceToHost);
		return count;
	}

	__global__ void removeSeam(unsigned char *input, unsigned char *output, unsigned int *dev_seams, size_t rows, size_t cols) {
		long index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= rows*cols*3)
			return;
		unsigned long thresh1 = 0;
		unsigned long thresh2 = 0*cols*3 + dev_seams[0]*3;
		for (int i = 0; i < rows; i++) {
			if (index >= thresh1 && index < thresh2)
				output[index - i * 3] = input[index];
			thresh1 = i*cols * 3 + dev_seams[i] * 3 + 3;
			thresh2 = (i + 1)*cols * 3 + dev_seams[(i + 1)] * 3;
		}
	}

	/*__global__ void removeSeam(unsigned char *input, unsigned char *output, unsigned long thresh1, unsigned long thresh2, size_t n, size_t row) {
		long index = (blockIdx.x * blockDim.x) + threadIdx.x;
		if (index >= n)
			return;
		if (index >= thresh1 && index < thresh2)
			output[index - row * 3] = input[index];
	}*/

	Mat gpuCarve(cv::Mat im, int direction, int seams) {
		unsigned char *dev_imgPtr, *dev_imgPtrBuffer;
		unsigned int *dev_energy;
		unsigned int *dev_seams;
		int *dev_boolean, *dev_indices;
		unsigned char *imgPtr = new unsigned char[im.rows * im.cols * im.channels()];

		if (direction == 0) {
			transpose(im, im);
		}
		unsigned char *cvPtr = im.ptr<unsigned char>(0);
		for (size_t i = 0; i < im.rows * im.cols * im.channels(); ++i) {
			imgPtr[i] = cvPtr[i];
		}

		int paddedArraySize = 1 << ilog2ceil(im.rows*im.cols*3);
		hipMalloc((void**)&dev_imgPtr, sizeof(unsigned char)*im.rows * im.cols * im.channels());
		hipMalloc((void**)&dev_imgPtrBuffer, sizeof(unsigned char)*im.rows * im.cols * im.channels());
		hipMalloc((void**)&dev_energy, sizeof(unsigned int)*im.rows * im.cols);
		hipMalloc((void**)&dev_seams, sizeof(unsigned int)*im.cols);
		hipMalloc((void**)&dev_boolean, paddedArraySize * sizeof(int));
		hipMalloc((void**)&dev_indices, paddedArraySize * sizeof(int));
		hipMemcpy(dev_imgPtr, imgPtr, sizeof(unsigned char)*im.rows*im.cols*im.channels(), hipMemcpyHostToDevice);

		size_t rows = im.rows;
		size_t cols = im.cols;
		if (direction == 0) {
			for (int i = 0; i < seams; ++i) {
				const dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y, 1);
				computeEnergyGPU << <gridSize, blockSize >> >(rows, cols, dev_imgPtr, dev_energy);
				unsigned int *energy = new unsigned int[rows*cols * 3];
				hipMemcpy(energy, dev_energy, sizeof(unsigned int)*rows*cols, hipMemcpyDeviceToHost);
				vector<uint> seam = findVerticalSeam(energy, rows, cols);
				hipMemcpy(dev_seams, &seam[0], sizeof(unsigned int)*rows, hipMemcpyHostToDevice);
				#if STREAMCOMPACTION
				removeVerticalSeamGPU(rows, cols, dev_imgPtr, dev_imgPtrBuffer, dev_seams, dev_boolean, dev_indices);
				#endif
				#if STREAMCOMPACTION!=1
				dim3 gridsize((rows*(cols)* 3 + BLOCK_SIZE - 1) / BLOCK_SIZE);
				removeSeam << <gridsize, BLOCK_SIZE >> >(dev_imgPtr, dev_imgPtrBuffer, dev_seams, rows, cols);
				hipMemcpy(dev_imgPtr, dev_imgPtrBuffer, sizeof(unsigned char)*rows*(cols)* 3, hipMemcpyDeviceToDevice);
				#endif
				cols--;
			}
		}
		// Vertical
		else {
			for (int i = 0; i < seams; ++i) {
				const dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y, 1);
				#if PROFILE
					GpuTimer timer;
					timer.Start();
				#endif
				computeEnergyGPU << <gridSize, blockSize >> >(rows, cols, dev_imgPtr, dev_energy);
				#if PROFILE
					timer.Stop();
					printf("Compute Energy, %f ,", timer.Elapsed());
				#endif

				unsigned int *energy = new unsigned int[rows*cols * 3];
				hipMemcpy(energy, dev_energy, sizeof(unsigned int)*rows*cols, hipMemcpyDeviceToHost);
				#if PROFILE
					CpuTimer timer2;
					timer2.Start();
				#endif
				vector<uint> seam = findVerticalSeam(energy, rows, cols);
				#if PROFILE
					timer2.Stop();
					printf("find seam, %f ,", timer2.Elapsed());
				#endif

				hipMemcpy(dev_seams, &seam[0], sizeof(unsigned int)*rows, hipMemcpyHostToDevice);
				#if STREAMCOMPACTION
				removeVerticalSeamGPU(rows, cols, dev_imgPtr, dev_imgPtrBuffer, dev_seams, dev_boolean, dev_indices);
				#endif
				#if STREAMCOMPACTION!=1
				#if PROFILE
				timer.Start();
				#endif
				dim3 gridsize((rows*(cols)* 3 + BLOCK_SIZE - 1) / BLOCK_SIZE);
				removeSeam << <gridsize, BLOCK_SIZE >> >(dev_imgPtr, dev_imgPtrBuffer, dev_seams, rows, cols);
				hipMemcpy(dev_imgPtr, dev_imgPtrBuffer, sizeof(unsigned char)*rows*(cols)* 3, hipMemcpyDeviceToDevice);
				#if PROFILE
				timer.Stop();
				printf("Remove Seam, %f\n", timer.Elapsed());
				#endif
				#endif
				cols--;
			}
		}
		hipMemcpy(imgPtr, dev_imgPtr, sizeof(unsigned char)*rows*(cols)* 3, hipMemcpyDeviceToHost);
		int sizes[2];
		sizes[0] = rows;
		sizes[1] = cols;
		cv::Mat carved(2, sizes, CV_8UC3, (void *)imgPtr);
		if (direction == 0) {
			transpose(carved, carved);
		}

		hipFree(&dev_imgPtr);
		hipFree(&dev_imgPtrBuffer);
		hipFree(&dev_energy);
		hipFree(&dev_seams);
		hipFree(&dev_boolean);
		hipFree(&dev_indices);

		return carved;

	}
}